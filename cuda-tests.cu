#include "cuda-tests.h"

#ifdef __NVCC__

int test16()
{
	hipError_t err = hipSuccess;
	int n = 1024;
	int size = sizeof(double) * n;

	double *h_vec = new double[n];
	double *d_vec = NULL;

	err = hipMalloc((void**)&d_vec, size);

	if (err != hipSuccess)
    {
        cout << "Failed to allocate device vector!" << endl;
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_vec, h_vec, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        cout << "Failed to copy vector A from host to device!" << endl;
        exit(EXIT_FAILURE);
    }

	int threads = 256;
	int blocks = n / threads;

	CUDA_Haar_Decomp <<< threads, blocks >>> (d_vec, n, true);

	hipDeviceSynchronize();

	err = hipMemcpy(h_vec, d_vec, size, hipMemcpyDeviceToHost);

	if (err != hipSuccess)
    {
        cout << "Failed to copy vector C from device to host!" << endl;
        exit(EXIT_FAILURE);
    }

    hipFree(d_vec);

    cout << h_vec[0] << endl;

    delete [] h_vec;

    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        cout << "Failed to deinitialize the device!" << endl;
        exit(EXIT_FAILURE);
    }

	return 0;
}

#endif
