#include "hip/hip_runtime.h"
#include "int-haar-cuda.h"

#ifdef __NVCC__

template <typename T>
__global__ void _CUDA_Haar_Normalization_Step(T *data, int n)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < n)
	{
		data[i] = data[i] / sqrt(float(n));
	}
}

template <typename T>
__global__ void _CUDA_Haar_DecompStep(T *data, T *result, int n, T div)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < n)
	{
		result[i]     = (data[2*i] + data[2*i + 1]) / div;
		result[i + n] = (data[2*i] - data[2*i + 1]) / div;
	}
}

template <typename T>
__global__ void _CUDA_Haar_Next_Data_Level(T *data, T *result, int n)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < n)
	{
		data[i] = result[i];
	}
}

template <typename T>
void _CUDA_Haar_Decomp(T *h_vec, int n, bool normal)
{
	hipError_t err = hipSuccess;
	T *d_vec = NULL;
	T *r_vec = NULL;
	int size = sizeof(T) * n;
	int threads = 256;
	int blocks;

	if (threads > n) threads = n;
	blocks = n / threads;

	err = hipMalloc((void**)&d_vec, size);

	if (err != hipSuccess)
	{
		cout << "Failed to allocate device vector" << endl;
		exit(EXIT_FAILURE);
	}

    err = hipMemcpy(d_vec, h_vec, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        cout << "Failed to copy vector A from host to device!" << endl;
        exit(EXIT_FAILURE);
    }

	if (normal)
	{
		_CUDA_Haar_Normalization_Step <<< blocks, threads >>> (d_vec, n);
		hipDeviceSynchronize();
	}

	int n_step = n;
	int size_step;
	T div;

	if (normal) div = sqrt(T(2.0));
	else        div = T(2.0);

	T *temp = new T[n];

	while (n_step > 1)
	{
		size_step = sizeof(T) * n_step;
		err = hipMalloc((void**)&r_vec, size_step);

		if (err != hipSuccess)
		{
			cout << "Failed to allocate result device vector!" << endl;
			exit(EXIT_FAILURE);
		}

		if (threads > n_step / 2) threads = n_step / 2;
		blocks = (n_step / 2) / threads;

		hipMemcpy(temp, d_vec, size, hipMemcpyDeviceToHost);

		cout << temp[0] << endl;
	    cout << temp[1] << endl;
	    cout << temp[2] << endl;
	    cout << temp[3] << endl;
	    cout << "-----" << endl;

		_CUDA_Haar_DecompStep <<< blocks, threads >>> (d_vec, r_vec, n_step/2, div);
		hipDeviceSynchronize();

		threads *= 2;

		if (threads > n) threads = n;

		_CUDA_Haar_Next_Data_Level <<< blocks, threads >>> (d_vec, r_vec, n_step);
		hipDeviceSynchronize();

		hipMemcpy(temp, d_vec, size, hipMemcpyDeviceToHost);

		cout << temp[0] << endl;
	    cout << temp[1] << endl;
	    cout << temp[2] << endl;
	    cout << temp[3] << endl;
	    cout << "-----" << endl;

		err = hipFree(r_vec);

		if (err != hipSuccess)
		{
			cout << "Failed to free result vector on device!" << endl;
			exit(EXIT_FAILURE);
		}

		n_step /= 2;
	}

	delete [] temp;

	err = hipMemcpy(h_vec, d_vec, size, hipMemcpyDeviceToHost);

	if (err != hipSuccess)
    {
        cout << "Failed to copy vector C from device to host!" << endl;
        exit(EXIT_FAILURE);
    }

    hipFree(d_vec);

    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        cout << "Failed to deinitialize the device!" << endl;
        exit(EXIT_FAILURE);
    }

}

void CUDA_Haar_Decomp(double *h_vec, int n, bool normal)
{
	_CUDA_Haar_Decomp(h_vec, n, normal);
}


#endif
